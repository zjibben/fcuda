
#include <hip/hip_runtime.h>
//#include <cuda_runtime.h>
//#include <stdio.h>

// device queries
extern "C" int ext_cudaGetDevice(int *device)
{
  return hipGetDevice(device);
}

extern "C" int ext_cudaGetDeviceCount(int *ndevices)
{
  return hipGetDeviceCount(ndevices);
}

extern "C" int ext_cudaGetDeviceProperties(hipDeviceProp_t *prop, int device)
{
  return hipGetDeviceProperties(prop, device);
}

// malloc
extern "C" int ext_cudaMalloc(void** devPtr, size_t size)
{
  return hipMalloc(devPtr, size);
}

extern "C" int ext_cudaFree(void* devPtr)
{
  return hipFree(devPtr);
}

// memcpy
extern "C" int ext_cudaMemcpy(void* dst, const void *src, size_t count, hipMemcpyKind kind)
{
  return hipMemcpy(dst, src, count, kind);
}
